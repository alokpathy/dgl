
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdint>
#include <hip/hip_runtime_api.h>

#define KB 1024
#define GB 1073741824

int main(int argc, char **argv) {

  float *dA, *dB;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  for (int64_t i = 1; i < 10000; i += 1000) {
    int64_t datasize = i * KB;
    hipMalloc(&dA, datasize);
    hipMalloc(&dB, datasize);

    // warmup memcpy
    hipMemcpyAsync(dB, dA, datasize, hipMemcpyDeviceToDevice);

    // timing memcpy
    hipEventRecord(start);
    hipMemcpyAsync(dB, dA, datasize, hipMemcpyDeviceToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float copytime = 0.0f;
    hipEventElapsedTime(&copytime, start, stop);
    copytime = copytime / 1000; // seconds

    std::cout << "datasize: " << datasize << " copy_time: " << copytime << " bandwidth GB/s: " << (datasize / copytime / GB) << "\n";

    hipFree(dA);
    hipFree(dB);
  }
}
