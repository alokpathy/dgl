#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2020 by Contributors
 * \file array/cuda/spmm.cu
 * \brief SPMM C APIs and definitions.
 */
#include <cutlass/gemm/device/gemm.h>
#include <dgl/array.h>
#include "./spmm.cuh"
#include "./ge_spmm.cuh"
#include "./functor.cuh"
#include "../../runtime/cuda/cuda_common.h"

// #define TIMING

namespace dgl {

using namespace cuda;

namespace aten {
namespace {

/*! \brief Call cuBLAS geam API for transpose operation for float and double. */
template <typename DType>
hipblasStatus_t Xgeam(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n,
    const DType* alpha, const DType* A, int lda,
    const DType* beta, const DType* B, int ldb,
    DType* C, int ldc) {
  LOG(INFO) << "Not supported dtype";
  return HIPBLAS_STATUS_EXECUTION_FAILED;
}

template <>
hipblasStatus_t Xgeam<float>(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n,
    const float* alpha, const float* A, int lda,
    const float* beta, const float* B, int ldb,
    float* C, int ldc) {
  return hipblasSgeam(handle, transa, transb, m, n, alpha, A, lda,
      beta, B, ldb, C, ldc);
}

template <>
hipblasStatus_t Xgeam<double>(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n,
    const double* alpha, const double* A, int lda,
    const double* beta, const double* B, int ldb,
    double* C, int ldc) {
  return hipblasDgeam(handle, transa, transb, m, n, alpha, A, lda,
      beta, B, ldb, C, ldc);
}

/* \brief IndexSelect operator kernel implementation.
 * \note duplicate of IndexSelectKernel defined in array_index_select.cu
 */
template <typename DType, typename IdType>
__global__ void _IndexSelectKernel(
    const DType* __restrict__ in,
    const IdType* __restrict__ idx,
    DType* __restrict__ out,
    int n, int m) {
  int i = blockIdx.x;
  for (int j = threadIdx.x; j < m; j += blockDim.x)
    out[i * m + j] = in[idx[i] * m + j];
}

/* \brief Transpose operator kernel implementation.
 * \note not efficient but it's not a bottleneck, used for float16 dtype.
 */
template <typename DType>
__global__ void _TransposeKernel(
    const DType* __restrict__ in,
    DType* __restrict__ out,
    int n, int m) {
  int i = blockIdx.x;
  for (int j = threadIdx.x; j < m; j += blockDim.x)
    out[i * m + j] = in[j * n + i];
}

/*
 * \brief Tranpose the input matrix.
 * \param row number of rows of input matrix.
 * \param col number of columns of input matrix.
 */
template <typename DType>
void _Transpose(const DType* in, DType* out,
                int row, int col) {
  DType alpha = 1., beta = 0.;
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  if (!thr_entry->cublas_handle)
    CUBLAS_CALL(hipblasCreate(&(thr_entry->cublas_handle)));
  CUBLAS_CALL(hipblasSetStream(thr_entry->cublas_handle, thr_entry->stream));
  CUBLAS_CALL(Xgeam<DType>(
      thr_entry->cublas_handle,
      HIPBLAS_OP_T,
      HIPBLAS_OP_N,
      row, col,
      &alpha, in, col,
      &beta, nullptr, row,
      out, row));
}

/*
 * \brief Tranpose the input matrix for data type half.
 * \note cuBLAS has no geam API for half data type, fallback to our kernel.
 */
template <>
void _Transpose<half>(const half* in, half* out,
                      int row, int col) {
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  int nt = FindNumThreads(row);
  int nb = col;
  CUDA_KERNEL_CALL(_TransposeKernel, nb, nt, 0, thr_entry->stream, in, out, col, row);
}

/*
 * \brief
 */
template <typename DType, typename IdType>
__global__ void _IndexSelectKernel(const DType* array, const IdType* index,
                                   int64_t length, DType* out) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_x = gridDim.x * blockDim.x;
  while (tx < length) {
    out[tx] = array[index[tx]];
    tx += stride_x;
  }
}

/* \brief IndexSelect operator.
 * \note duplicate of IndexSelect defined in array_op.h but it can
 *    not be applied to float16 dtype.
 */
template<typename DType, typename IdType>
NDArray _IndexSelect(NDArray array, NDArray index) {
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  const DType* array_data = static_cast<DType*>(array->data);
  const IdType* idx_data = static_cast<IdType*>(index->data);
  const int64_t arr_len = array->shape[0];
  const int64_t len = index->shape[0];
  NDArray ret = NDArray::Empty({len}, array->dtype, array->ctx);
  if (len == 0)
    return ret;
  DType* ret_data = static_cast<DType*>(ret->data);
  const int nt = FindNumThreads(len);
  const int nb = (len + nt - 1) / nt;
  CUDA_KERNEL_CALL(_IndexSelectKernel, nb, nt, 0, thr_entry->stream,
      array_data, idx_data, len, ret_data);
  return ret;
}

}  // namespace

namespace cusparse {

#if CUDART_VERSION < 11000
template <typename DType>
hipsparseStatus_t Xcsrmm2(hipsparseHandle_t handle, hipsparseOperation_t transA,
    hipsparseOperation_t transB, int m, int n, int k, int nnz,
    const DType* alpha, const hipsparseMatDescr_t descrA,
    const DType* csrValA, const int* csrRowPtrA, const int* csrColIndA,
    const DType* B, int ldb, const DType* beta, DType* C, int ldc) {
  LOG(INFO) << "Not supported dtype";
  return HIPSPARSE_STATUS_EXECUTION_FAILED;
}

template <>
hipsparseStatus_t Xcsrmm2<float>(hipsparseHandle_t handle, hipsparseOperation_t transA,
    hipsparseOperation_t transB, int m, int n, int k, int nnz,
    const float* alpha, const hipsparseMatDescr_t descrA,
    const float* csrValA, const int* csrRowPtrA, const int* csrColIndA,
    const float* B, int ldb, const float* beta, float* C, int ldc) {
  return hipsparseScsrmm2(handle, transA, transB, m, n, k, nnz,
      alpha, descrA, csrValA, csrRowPtrA, csrColIndA,
      B, ldb, beta, C, ldc);
}

template <>
hipsparseStatus_t Xcsrmm2<double>(hipsparseHandle_t handle, hipsparseOperation_t transA,
    hipsparseOperation_t transB, int m, int n, int k, int nnz,
    const double* alpha, const hipsparseMatDescr_t descrA,
    const double* csrValA, const int* csrRowPtrA, const int* csrColIndA,
    const double* B, int ldb, const double* beta, double* C, int ldc) {
  return hipsparseDcsrmm2(handle, transA, transB, m, n, k, nnz,
      alpha, descrA, csrValA, csrRowPtrA, csrColIndA,
      B, ldb, beta, C, ldc);
}
#endif

/*! Cusparse implementation of SpMM on Csr format. */
template <typename DType, typename IdType>
void CusparseCsrmm2(
    const DLContext& ctx,
    const CSRMatrix& csr,
    const DType* B_data, const DType* A_data,
    DType* C_data,
    int x_length) {
  // We use csrmm2 to perform following operation:
  // C = A x B, where A is a sparse matrix in csr format, B is the dense matrix for node
  // feature tensor. However, since cusparse only supports column-major, while our tensor
  // is stored in row-major, the actual computation is:
  // C = trans(A x trans(B)).
  // Currently, we use cublasXgeam to implement transposition and allocate intermediate
  // workspace memory for this.
  const int m = csr.num_rows;
  const int n = x_length;
  const int k = csr.num_cols;
  const int nnz = csr.indices->shape[0];
  const DType alpha = 1.0;
  const DType beta = 0.0;
  // device
  auto device = runtime::DeviceAPI::Get(ctx);
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  // allocate cusparse handle if needed
  if (!thr_entry->cusparse_handle) {
    CUSPARSE_CALL(hipsparseCreate(&(thr_entry->cusparse_handle)));
  }
  CUSPARSE_CALL(hipsparseSetStream(thr_entry->cusparse_handle, thr_entry->stream));
  // all one data array
  DType* valptr = nullptr;
  if (!A_data) {
    valptr = static_cast<DType*>(device->AllocWorkspace(ctx, nnz * sizeof(DType)));
    _Fill(valptr, nnz, static_cast<DType>(1.));
  }
#if CUDART_VERSION >= 11000
  hipsparseSpMatDescr_t matA;
  hipsparseDnMatDescr_t matB, matC;
  constexpr auto dtype = cuda_dtype<DType>::value;
  constexpr auto idtype = cusparse_idtype<IdType>::value;
  CUSPARSE_CALL(hipsparseCreateCsr(&matA,
      m, k, nnz,
      static_cast<IdType*>(csr.indptr->data),
      static_cast<IdType*>(csr.indices->data),
      const_cast<DType*>(valptr? valptr : A_data),
      idtype, idtype,
      HIPSPARSE_INDEX_BASE_ZERO, dtype));
  CUSPARSE_CALL(hipsparseCreateDnMat(&matB,
      k, n, n,
      const_cast<DType*>(B_data), dtype, HIPSPARSE_ORDER_ROW));
  CUSPARSE_CALL(hipsparseCreateDnMat(&matC,
      m, n, n,
      C_data, dtype, HIPSPARSE_ORDER_ROW));

  auto transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  auto transB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  size_t workspace_size;
  CUSPARSE_CALL(hipsparseSpMM_bufferSize(
      thr_entry->cusparse_handle, transA, transB,
      &alpha, matA, matB, &beta, matC,
      dtype, HIPSPARSE_SPMM_CSR_ALG2,
      &workspace_size));
  void* workspace = device->AllocWorkspace(ctx, workspace_size);
  CUSPARSE_CALL(hipsparseSpMM(
      thr_entry->cusparse_handle, transA, transB,
      &alpha, matA, matB, &beta, matC,
      dtype, HIPSPARSE_SPMM_CSR_ALG2,
      workspace));
  device->FreeWorkspace(ctx, workspace);

  CUSPARSE_CALL(hipsparseDestroySpMat(matA));
  CUSPARSE_CALL(hipsparseDestroyDnMat(matB));
  CUSPARSE_CALL(hipsparseDestroyDnMat(matC));
#else
  // allocate matrix for temporary transposed output
  DType* trans_out = static_cast<DType*>(device->AllocWorkspace(ctx, m * n * sizeof(DType)));

  hipsparseMatDescr_t descr;
  CUSPARSE_CALL(hipsparseCreateMatDescr(&descr));
  CUSPARSE_CALL(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
  CUSPARSE_CALL(hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO));
  CUSPARSE_CALL(Xcsrmm2<DType>(
      thr_entry->cusparse_handle,
      HIPSPARSE_OPERATION_NON_TRANSPOSE,
      HIPSPARSE_OPERATION_TRANSPOSE,
      m, n, k, nnz, &alpha,
      descr, (valptr)? valptr : A_data,
      static_cast<int32_t*>(csr.indptr->data),
      static_cast<int32_t*>(csr.indices->data),
      B_data, n, &beta, trans_out, m));
  CUSPARSE_CALL(hipsparseDestroyMatDescr(descr));
  // transpose the output matrix
  _Transpose(trans_out, C_data, n, m);
  device->FreeWorkspace(ctx, trans_out);
#endif
  if (valptr)
    device->FreeWorkspace(ctx, valptr);
}
}  // namespace cusparse

#define SWITCH_OP(op, Op, ...)                                      \
  do {                                                              \
    if ((op) == "add") {                                            \
      typedef cuda::binary::Add<DType> Op;                          \
      { __VA_ARGS__ }                                               \
    } else if ((op) == "sub") {                                     \
      typedef cuda::binary::Sub<DType> Op;                          \
      { __VA_ARGS__ }                                               \
    } else if ((op) == "mul") {                                     \
      typedef cuda::binary::Mul<DType> Op;                          \
      { __VA_ARGS__ }                                               \
    } else if ((op) == "div") {                                     \
      typedef cuda::binary::Div<DType> Op;                          \
      { __VA_ARGS__ }                                               \
    } else if ((op) == "copy_lhs") {                                \
      typedef cuda::binary::CopyLhs<DType> Op;                      \
      { __VA_ARGS__ }                                               \
    } else if ((op) == "copy_rhs") {                                \
      typedef cuda::binary::CopyRhs<DType> Op;                      \
      { __VA_ARGS__ }                                               \
    } else {                                                        \
      LOG(FATAL) << "Unsupported SpMM binary operator: " << op;     \
    }                                                               \
  } while (0)

/*!
 * \brief Determine whether cusparse SpMM function is applicable.
 */
template <int bits, typename IdType>
inline bool cusparse_available() {
#if CUDART_VERSION < 11000
  if (std::is_same<IdType, int>::value)
    if (bits > 16)
      return true;
  return false;
#else
  if (bits == 16)
    return false;  // cusparse's SpMM on fp16 is slow, temporally disabled.
  return true;
#endif
}

__global__ void SetCsrOffsets(int *offsets, int M1, int N1, int M2, int N2) {
  int     id = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  
  for (int i = id; i < M1; i += stride) {
    offsets[i] = i * N1;
  }

  for (int i = id + M1; i < M1 + M2 + 1; i += stride) {
    offsets[i] = M1 * N1 + (i - M1) * N2;
  }
}

__global__ void SetCsrColumns(int *columns, int M1, int N1, int M2, int N2) {
  int     id = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  
  for (int i = id; i < M1 * N1; i += stride) {
    columns[i] = i % N1;
  }

  for (int i = id + M1 * N1; i < M1 * N1 + M2 * N2; i += stride) {
    columns[i] = N1 + (i - M1 * N1) % N2;
  }
}

// TODO: template this with generic DType
void fused_gemm(NDArray A1, NDArray B1, NDArray C1, int M1, int K1, int N1,
                    NDArray A2, NDArray B2, NDArray C2, int M2, int K2, int N2) {

  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();

#ifdef TIMING
  hipEvent_t total_start, total_stop;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventCreate(&total_start);
  hipEventCreate(&total_stop);

  hipEventRecord(total_start);
#endif
  hipsparseSpMatDescr_t matA, matB, matC;
  void*  dBuffer1    = NULL, *dBuffer2   = NULL;
  size_t bufferSize1 = 0,    bufferSize2 = 0;

  // allocate cusparse handle if needed
  if (!thr_entry->cusparse_handle) {
    CUSPARSE_CALL(hipsparseCreate(&(thr_entry->cusparse_handle)));
  }

#ifdef TIMING
  hipEventRecord(start);
#endif
  // Convert A into sparse matrix
  int *dA_csrOffsets, *dA_columns;
  float *dA_values;
  // CUDA_CALL( hipMalloc(&dA_csrOffsets, (M + 1) * sizeof(int)) );
  // CUDA_CALL( hipMalloc(&dA_columns, M * K * sizeof(int)) );
  int matA_numrows = M1 + M2;
  int matA_nnz = M1 * K1 + M2 * K2;
  CUDA_CALL( hipMalloc(&dA_csrOffsets, (matA_numrows + 1) * sizeof(int)) );
  CUDA_CALL( hipMalloc(&dA_columns, matA_nnz * sizeof(int)) );
  CUDA_CALL( hipMalloc(&dA_values, matA_nnz * sizeof(float)) );

  const int nt_aoff = FindNumThreads(matA_numrows + 1);
  const int nb_aoff = (matA_numrows + 1 + nt_aoff - 1) / nt_aoff;

  const int nt_acol = FindNumThreads(matA_nnz);
  const int nb_acol = (matA_nnz + nt_acol - 1) / nt_acol;

  // CUDA_KERNEL_CALL( SetCsrOffsets, nb_aoff, nt_aoff, 0, thr_entry->stream, dA_csrOffsets, M, K );
  // CUDA_KERNEL_CALL( SetCsrColumns, nb_acol, nt_acol, 0, thr_entry->stream, dA_columns, M, K );
  // CUSPARSE_CALL( hipsparseCreateCsr(&matA, M, K, M * K,
  //                                   dA_csrOffsets, dA_columns, A->data,
  //                                   HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
  //                                   HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) );
  CUDA_KERNEL_CALL( SetCsrOffsets, nb_aoff, nt_aoff, 0, thr_entry->stream, dA_csrOffsets, M1, K1, M2, K2 );
  CUDA_KERNEL_CALL( SetCsrColumns, nb_acol, nt_acol, 0, thr_entry->stream, dA_columns, M1, K1, M2, K2 );
  CUDA_CALL( hipMemcpy(dA_values, A1->data, M1 * K1 * sizeof(float), hipMemcpyDeviceToDevice) );
  CUDA_CALL( hipMemcpy(dA_values + M1 * K1, A2->data, M2 * K2 * sizeof(float), hipMemcpyDeviceToDevice) );

  CUSPARSE_CALL( hipsparseCreateCsr(&matA, M1 + M2, K1 + K2, matA_nnz,
                                    dA_csrOffsets, dA_columns, dA_values,
                                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) );

  // Convert B into sparse matrix
  int *dB_csrOffsets, *dB_columns;
  float *dB_values;
  // CUDA_CALL( hipMalloc(&dB_csrOffsets, (K + 1) * sizeof(int)) );
  // CUDA_CALL( hipMalloc(&dB_columns, K * N * sizeof(int)) );
  int matB_numrows = K1 + K2;
  int matB_nnz = K1 * N1 + K2 * N2;
  CUDA_CALL( hipMalloc(&dB_csrOffsets, (matB_numrows + 1) * sizeof(int)) );
  CUDA_CALL( hipMalloc(&dB_columns, matB_nnz * sizeof(int)) );
  CUDA_CALL( hipMalloc(&dB_values, matB_nnz * sizeof(float)) );

  const int nt_boff = FindNumThreads(matB_numrows + 1);
  const int nb_boff = (matB_numrows + 1 + nt_boff - 1) / nt_boff;

  const int nt_bcol = FindNumThreads(matB_nnz);
  const int nb_bcol = (matB_nnz + nt_bcol - 1) / nt_bcol;

  // CUDA_KERNEL_CALL( SetCsrOffsets, nb_boff, nt_boff, 0, thr_entry->stream, dB_csrOffsets, K, N );
  // CUDA_KERNEL_CALL( SetCsrColumns, nb_bcol, nt_bcol, 0, thr_entry->stream, dB_columns, K, N );
  // CUSPARSE_CALL( hipsparseCreateCsr(&matB, K, N, K * N,
  //                                   dB_csrOffsets, dB_columns, B->data,
  //                                   HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
  //                                   HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) );
  CUDA_KERNEL_CALL( SetCsrOffsets, nb_boff, nt_boff, 0, thr_entry->stream, dB_csrOffsets, K1, N1, K2, N2 );
  CUDA_KERNEL_CALL( SetCsrColumns, nb_bcol, nt_bcol, 0, thr_entry->stream, dB_columns, K1, N1, K2, N2 );
  CUDA_CALL( hipMemcpy(dB_values, B1->data, K1 * N1 * sizeof(float), hipMemcpyDeviceToDevice) );
  CUDA_CALL( hipMemcpy(dB_values + K1 * N1, B2->data, K2 * N2 * sizeof(float), hipMemcpyDeviceToDevice) );
  CUSPARSE_CALL( hipsparseCreateCsr(&matB, K1 + K2, N1 + N2, matB_nnz,
                                    dB_csrOffsets, dB_columns, dB_values,
                                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) );

  // Convert C into sparse matrix
  // CUSPARSE_CALL( hipsparseCreateCsr(&matC, M, N, 0,
  //                                   NULL, NULL, NULL,
  //                                   HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
  //                                   HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) );
  CUSPARSE_CALL( hipsparseCreateCsr(&matC, M1 + M2, N1 + N2, 0,
                                    NULL, NULL, NULL,
                                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) );

  hipsparseSpGEMMDescr_t spgemmDesc;
  CUSPARSE_CALL( hipsparseSpGEMM_createDescr(&spgemmDesc) );

  // ask bufferSize1 bytes for external memory
  float               alpha       = 1.0f;
  float               beta        = 0.0f;
  hipsparseOperation_t opA         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  hipsparseOperation_t opB         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  CUSPARSE_CALL( hipsparseSpGEMM_workEstimation(thr_entry->cusparse_handle, opA, opB,
                                        &alpha, matA, matB, &beta, matC,
                                        HIP_R_32F, HIPSPARSE_SPGEMM_DEFAULT,
                                        spgemmDesc, &bufferSize1, NULL) );
  CUDA_CALL( hipMalloc((void**) &dBuffer1, bufferSize1) );

  // inspect the matrices A and B to understand the memory requirement for
  // the next step
  CUSPARSE_CALL( hipsparseSpGEMM_workEstimation(thr_entry->cusparse_handle, opA, opB,
                                        &alpha, matA, matB, &beta, matC,
                                        HIP_R_32F, HIPSPARSE_SPGEMM_DEFAULT,
                                        spgemmDesc, &bufferSize1, dBuffer1) );

  // ask bufferSize2 bytes for external memory
  CUSPARSE_CALL( hipsparseSpGEMM_compute(thr_entry->cusparse_handle, opA, opB,
                                 &alpha, matA, matB, &beta, matC,
                                 HIP_R_32F, HIPSPARSE_SPGEMM_DEFAULT,
                                 spgemmDesc, &bufferSize2, NULL) );
  CUDA_CALL( hipMalloc((void**) &dBuffer2, bufferSize2) );

#ifdef TIMING
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float spgemm_preproc = 0;
  hipEventElapsedTime(&spgemm_preproc, start, stop);

  hipEventRecord(start);
#endif
  // compute the intermediate product of A * B
  CUSPARSE_CALL( hipsparseSpGEMM_compute(thr_entry->cusparse_handle, opA, opB,
                                             &alpha, matA, matB, &beta, matC,
                                         HIP_R_32F, HIPSPARSE_SPGEMM_DEFAULT,
                                         spgemmDesc, &bufferSize2, dBuffer2) );
#ifdef TIMING
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float spgemm_compute = 0;
  hipEventElapsedTime(&spgemm_compute, start, stop);

  hipEventRecord(start);
#endif
  // allocate matrix C
  int *dC_csrOffsets, *dC_columns;
  float *dC_values;
  // CUDA_CALL( hipMalloc((void**) &dC_csrOffsets, (M + 1) * sizeof(int)) );
  // CUDA_CALL( hipMalloc((void**) &dC_columns, M * N * sizeof(int))   ); 
  int matC_numrows = M1 + M2;
  int matC_nnz = M1 * N1 + M2 * N2;
  CUDA_CALL( hipMalloc((void**) &dC_csrOffsets, (matC_numrows + 1) * sizeof(int)) );
  CUDA_CALL( hipMalloc((void**) &dC_columns, matC_nnz * sizeof(int))   ); 
  CUDA_CALL( hipMalloc((void**) &dC_values, matC_nnz * sizeof(float))   ); 

  // update matC with the new pointers
  // CUSPARSE_CALL( hipsparseCsrSetPointers(matC, dC_csrOffsets, dC_columns, C->data) );
  CUSPARSE_CALL( hipsparseCsrSetPointers(matC, dC_csrOffsets, dC_columns, dC_values) );

  // if beta != 0, hipsparseSpGEMM_copy reuses/updates the values of dC_values

  // copy the final products to the matrix C
  CUSPARSE_CALL( hipsparseSpGEMM_copy(thr_entry->cusparse_handle, opA, opB,
                          &alpha, matA, matB, &beta, matC,
                          HIP_R_32F, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc) );

  // set C1 and C2's data to dC_values
  // Q: could we just set C1->data and C2->data pointers instead of a memcpy, or would that be a mem leak?
  CUDA_CALL( hipMemcpy(C1->data, dC_values, M1 * N1 * sizeof(float), hipMemcpyDeviceToDevice) );
  CUDA_CALL( hipMemcpy(C2->data, dC_values + (M1 * N1), M2 * N2 * sizeof(float), hipMemcpyDeviceToDevice) );

#ifdef TIMING
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float spgemm_copy = 0;
  hipEventElapsedTime(&spgemm_copy, start, stop);

  hipEventRecord(start);
#endif
  // destroy matrix/vector descriptors
  CUDA_CALL( hipFree(dA_csrOffsets) );
  CUDA_CALL( hipFree(dA_columns) );
  CUDA_CALL( hipFree(dA_values) );
  CUDA_CALL( hipFree(dB_csrOffsets) );
  CUDA_CALL( hipFree(dB_columns) );
  CUDA_CALL( hipFree(dB_values) );
  CUDA_CALL( hipFree(dC_csrOffsets) );
  CUDA_CALL( hipFree(dC_columns) );
  CUDA_CALL( hipFree(dC_values) );
  CUDA_CALL( hipFree(dBuffer1) );
  CUDA_CALL( hipFree(dBuffer2) );

  CUSPARSE_CALL( hipsparseSpGEMM_destroyDescr(spgemmDesc) );
  CUSPARSE_CALL( hipsparseDestroySpMat(matA) );
  CUSPARSE_CALL( hipsparseDestroySpMat(matB) );
  CUSPARSE_CALL( hipsparseDestroySpMat(matC) );

#ifdef TIMING
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float spgemm_destroy = 0;
  hipEventElapsedTime(&spgemm_destroy, start, stop);

  hipEventRecord(total_stop);
  hipEventSynchronize(total_stop);
  float spgemm_total = 0;
  hipEventElapsedTime(&spgemm_total, total_start, total_stop);
  
  float spgemm_accum_time = spgemm_preproc + spgemm_compute + spgemm_copy + spgemm_destroy;
  printf("spgemm_total: %f %f\n", spgemm_total, spgemm_accum_time / spgemm_total);
  printf("spgemm_preproc: %f\n", spgemm_preproc);
  printf("spgemm_compute: %f\n", spgemm_compute);
  printf("spgemm_copy: %f\n", spgemm_copy);
  printf("spgemm_destroy: %f\n", spgemm_destroy); fflush(stdout);
#endif
}

/*!
 * \brief CUDA implementation of g-SpMM on Csr format.
 * \note use cusparse if the reduce operator is `sum` and there is
 *       no broadcast, use dgl's kernel in other cases.
 */
template <int XPU, typename IdType, int bits>
void SpMMCsr(const std::string& op, const std::string& reduce,
             const BcastOff& bcast,
             const CSRMatrix& csr,
             NDArray ufeat,
             NDArray efeat,
             NDArray out,
             std::vector<NDArray> out_aux) {
  int64_t feat_len = bcast.out_len;
  bool is_scalar_efeat = efeat.NumElements() == csr.indices->shape[0];
  bool use_efeat = op != "copy_lhs";

  if (reduce == "sum") {
    if (op == "copy_lhs" && cusparse_available<bits, IdType>()) {  // cusparse
      int64_t x_length = 1;
      for (int i = 1; i < ufeat->ndim; ++i)
        x_length *= ufeat->shape[i];
      SWITCH_BITS(bits, DType, {
        cusparse::CusparseCsrmm2<DType, IdType>(
            ufeat->ctx, csr,
            static_cast<DType*>(ufeat->data),
            nullptr,
            static_cast<DType*>(out->data),
            x_length);
      });
    } else if (op == "mul" && is_scalar_efeat && cusparse_available<bits, IdType>()) {  // cusparse
      int64_t x_length = 1;
      for (int i = 1; i < ufeat->ndim; ++i)
        x_length *= ufeat->shape[i];
      if (!IsNullArray(csr.data)) {
        SWITCH_BITS(bits, DType, {
          efeat = _IndexSelect<DType, IdType>(efeat, csr.data);
        });
      }
      SWITCH_BITS(bits, DType, {
        cusparse::CusparseCsrmm2<DType, IdType>(
            ufeat->ctx, csr,
            static_cast<DType*>(ufeat->data),
            static_cast<DType*>(efeat->data),
            static_cast<DType*>(out->data),
            x_length);
      });
    } else {  // general kernel
      SWITCH_BITS(bits, DType, {
        SWITCH_OP(op, Op, {
          cuda::SpMMCsr<IdType, DType, Op, cuda::reduce::Sum<IdType, DType> >(
              bcast, csr, ufeat, efeat, out, NullArray(), NullArray());
        });
      });
    }
  } else if (reduce == "max") {
    SWITCH_BITS(bits, DType, {
      SWITCH_OP(op, Op, {
        cuda::SpMMCsr<IdType, DType, Op, cuda::reduce::Max<IdType, DType> >(
            bcast, csr, ufeat, efeat, out, out_aux[0], out_aux[1]);
      });
    });
  } else if (reduce == "min") {
    SWITCH_BITS(bits, DType, {
      SWITCH_OP(op, Op, {
        cuda::SpMMCsr<IdType, DType, Op, cuda::reduce::Min<IdType, DType> >(
            bcast, csr, ufeat, efeat, out, out_aux[0], out_aux[1]);
      });
    });
  } else {
    LOG(FATAL) << "Not implemented";
  }
}


/*!
 * \brief CUDA implementation of g-SpMM on Coo format.
 */
template <int XPU, typename IdType, int bits>
void SpMMCoo(const std::string& op, const std::string& reduce,
             const BcastOff& bcast,
             const COOMatrix& coo,
             NDArray ufeat,
             NDArray efeat,
             NDArray out,
             std::vector<NDArray> out_aux) {
  
  if (reduce == "sum") {
    SWITCH_BITS(bits, DType, {
      SWITCH_OP(op, Op, {
        cuda::SpMMCoo<IdType, DType, Op, cuda::reduce::Sum<IdType, DType, true> > (
            bcast, coo, ufeat, efeat, out, NullArray(), NullArray());
      });
    });
  } else if (reduce == "max") {
    SWITCH_BITS(bits, DType, {
      SWITCH_OP(op, Op, {
        cuda::SpMMCoo<IdType, DType, Op, cuda::reduce::Max<IdType, DType, true> > (
            bcast, coo, ufeat, efeat, out, out_aux[0], out_aux[1]);
      });
    });
  }  else if (reduce == "min") {
    SWITCH_BITS(bits, DType, {
      SWITCH_OP(op, Op, {
        cuda::SpMMCoo<IdType, DType, Op, cuda::reduce::Min<IdType, DType, true> > (
            bcast, coo, ufeat, efeat, out, out_aux[0], out_aux[1]);
      });
    });
  } else {
    LOG(FATAL) << "Not implemented";
  }
}

template void SpMMCsr<kDLGPU, int32_t, 16>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const CSRMatrix& csr,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCsr<kDLGPU, int64_t, 16>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const CSRMatrix& csr,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCsr<kDLGPU, int32_t, 32>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const CSRMatrix& csr,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCsr<kDLGPU, int64_t, 32>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const CSRMatrix& csr,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCsr<kDLGPU, int32_t, 64>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const CSRMatrix& csr,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCsr<kDLGPU, int64_t, 64>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const CSRMatrix& csr,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);

template void SpMMCoo<kDLGPU, int32_t, 16>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const COOMatrix& coo,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCoo<kDLGPU, int64_t, 16>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const COOMatrix& coo,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCoo<kDLGPU, int32_t, 32>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const COOMatrix& coo,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCoo<kDLGPU, int64_t, 32>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const COOMatrix& coo,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCoo<kDLGPU, int32_t, 64>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const COOMatrix& coo,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCoo<kDLGPU, int64_t, 64>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const COOMatrix& coo,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);


}  // namespace aten
}  // namespace dgl
